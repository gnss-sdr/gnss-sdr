#include "hip/hip_runtime.h"
/*!
 * \file cuda_multicorrelator.cu
 * \brief High optimized CUDA GPU vector multiTAP correlator class
 * \authors <ul>
 *          <li> Javier Arribas, 2015. jarribas(at)cttc.es
 *          </ul>
 *
 * Class that implements a high optimized vector multiTAP correlator class for NVIDIA CUDA GPUs
 *
 * -------------------------------------------------------------------------
 *
 * Copyright (C) 2010-2018  (see AUTHORS file for a list of contributors)
 *
 * GNSS-SDR is a software defined Global Navigation
 *          Satellite Systems receiver
 *
 * This file is part of GNSS-SDR.
 *
 * GNSS-SDR is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * GNSS-SDR is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with GNSS-SDR. If not, see <https://www.gnu.org/licenses/>.
 *
 * -------------------------------------------------------------------------
 */

#include "cuda_multicorrelator.h"

#include <stdio.h>
#include <iostream>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#define ACCUM_N 128

__global__ void Doppler_wippe_scalarProdGPUCPXxN_shifts_chips(
    GPU_Complex *d_corr_out,
    GPU_Complex *d_sig_in,
    GPU_Complex *d_sig_wiped,
    GPU_Complex *d_local_code_in,
    float *d_shifts_chips,
    int code_length_chips,
    float code_phase_step_chips,
    float rem_code_phase_chips,
    int vectorN,
    int elementN,
    float rem_carrier_phase_in_rad,
    float phase_step_rad
)
{
    //Accumulators cache
    __shared__ GPU_Complex accumResult[ACCUM_N];

	// CUDA version of floating point NCO and vector dot product integrated
    float sin;
    float cos;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < elementN;
         i += blockDim.x * gridDim.x)
    {
    	__sincosf(rem_carrier_phase_in_rad + i*phase_step_rad, &sin, &cos);
    	d_sig_wiped[i] =  d_sig_in[i] * GPU_Complex(cos,-sin);
    }

    __syncthreads();
    ////////////////////////////////////////////////////////////////////////////
    // Cycle through every pair of vectors,
    // taking into account that vector counts can be different
    // from total number of thread blocks
    ////////////////////////////////////////////////////////////////////////////
    for (int vec = blockIdx.x; vec < vectorN; vec += gridDim.x)
    {
        //int vectorBase = IMUL(elementN, vec);
        //int vectorEnd  = elementN;

        ////////////////////////////////////////////////////////////////////////
        // Each accumulator cycles through vectors with
        // stride equal to number of total number of accumulators ACCUM_N
        // At this stage ACCUM_N is only preferred be a multiple of warp size
        // to meet memory coalescing alignment constraints.
        ////////////////////////////////////////////////////////////////////////
        for (int iAccum = threadIdx.x; iAccum < ACCUM_N; iAccum += blockDim.x)
        {
        	GPU_Complex sum = GPU_Complex(0,0);
            float local_code_chip_index=0.0;;
            //float code_phase;
            for (int pos = iAccum; pos < elementN; pos += ACCUM_N)
            {
            	//original sample code
                //sum = sum + d_sig_in[pos-vectorBase] * d_nco_in[pos-vectorBase] * d_local_codes_in[pos];
            	//sum = sum + d_sig_in[pos-vectorBase] * d_local_codes_in[pos];
            	//sum.multiply_acc(d_sig_in[pos],d_local_codes_in[pos+d_shifts_samples[vec]]);

            	//custom code for multitap correlator
            	// 1.resample local code for the current shift

            	local_code_chip_index= fmodf(code_phase_step_chips*__int2float_rd(pos)+ d_shifts_chips[vec] - rem_code_phase_chips, code_length_chips);

            	//Take into account that in multitap correlators, the shifts can be negative!
            	if (local_code_chip_index<0.0) local_code_chip_index+=(code_length_chips-1);
            	//printf("vec= %i, pos %i, chip_idx=%i chip_shift=%f \r\n",vec, pos,__float2int_rd(local_code_chip_index),local_code_chip_index);
            	// 2.correlate
            	sum.multiply_acc(d_sig_wiped[pos],d_local_code_in[__float2int_rd(local_code_chip_index)]);

            }
            accumResult[iAccum] = sum;
        }

        ////////////////////////////////////////////////////////////////////////
        // Perform tree-like reduction of accumulators' results.
        // ACCUM_N has to be power of two at this stage
        ////////////////////////////////////////////////////////////////////////
        for (int stride = ACCUM_N / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();

            for (int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x)
            {
                accumResult[iAccum] += accumResult[stride + iAccum];
            }
        }

        if (threadIdx.x == 0)
        	{
        		d_corr_out[vec] = accumResult[0];
        	}
    }
}

bool cuda_multicorrelator::init_cuda_integrated_resampler(
		int signal_length_samples,
		int code_length_chips,
		int n_correlators
		)
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
//	findCudaDevice(argc, (const char **)argv);
      hipDeviceProp_t  prop;
    int num_devices, device;
    hipGetDeviceCount(&num_devices);
    if (num_devices > 1) {
          int max_multiprocessors = 0, max_device = 0;
          for (device = 0; device < num_devices; device++) {
                  hipDeviceProp_t properties;
                  hipGetDeviceProperties(&properties, device);
                  if (max_multiprocessors < properties.multiProcessorCount) {
                          max_multiprocessors = properties.multiProcessorCount;
                          max_device = device;
                  }
                  printf("Found GPU device # %i\n",device);
          }
          //hipSetDevice(max_device);

          //set random device!
	  selected_gps_device=rand() % num_devices;//generates a random number between 0 and num_devices to split the threads between GPUs
          hipSetDevice(selected_gps_device); 

          hipGetDeviceProperties( &prop, max_device );
          //debug code
          if (prop.canMapHostMemory != 1) {
              printf( "Device can not map memory.\n" );
          }
          printf("L2 Cache size= %u \n",prop.l2CacheSize);
          printf("maxThreadsPerBlock= %u \n",prop.maxThreadsPerBlock);
          printf("maxGridSize= %i \n",prop.maxGridSize[0]);
          printf("sharedMemPerBlock= %lu \n",prop.sharedMemPerBlock);
          printf("deviceOverlap= %i \n",prop.deviceOverlap);
  	    printf("multiProcessorCount= %i \n",prop.multiProcessorCount);
    }else{
    	    hipGetDevice( &selected_gps_device);
    	    hipGetDeviceProperties( &prop, selected_gps_device );
    	    //debug code
    	    if (prop.canMapHostMemory != 1) {
    	        printf( "Device can not map memory.\n" );
    	    }

    	    printf("L2 Cache size= %u \n",prop.l2CacheSize);
    	    printf("maxThreadsPerBlock= %u \n",prop.maxThreadsPerBlock);
    	    printf("maxGridSize= %i \n",prop.maxGridSize[0]);
    	    printf("sharedMemPerBlock= %lu \n",prop.sharedMemPerBlock);
    	    printf("deviceOverlap= %i \n",prop.deviceOverlap);
    	    printf("multiProcessorCount= %i \n",prop.multiProcessorCount);
    }

	// (hipFuncSetCacheConfig(reinterpret_cast<const void*>(CUDA_32fc_x2_multiply_x2_dot_prod_32fc_), hipFuncCachePreferShared));

    // ALLOCATE GPU MEMORY FOR INPUT/OUTPUT and INTERNAL vectors
    size_t size = signal_length_samples * sizeof(GPU_Complex);

	//********* ZERO COPY VERSION ************
	// Set flag to enable zero copy access
    // Optimal in shared memory devices (like Jetson K1)
	//hipSetDeviceFlags(hipDeviceMapHost);

	//******** CudaMalloc version ***********

	// input signal GPU memory (can be mapped to CPU memory in shared memory devices!)
	//	hipMalloc((void **)&d_sig_in, size);
	//	hipMemset(d_sig_in,0,size);

	// Doppler-free signal (internal GPU memory)
	hipMalloc((void **)&d_sig_doppler_wiped, size);
	hipMemset(d_sig_doppler_wiped,0,size);

	// Local code GPU memory (can be mapped to CPU memory in shared memory devices!)
	hipMalloc((void **)&d_local_codes_in, sizeof(std::complex<float>)*code_length_chips);
	hipMemset(d_local_codes_in,0,sizeof(std::complex<float>)*code_length_chips);

    d_code_length_chips=code_length_chips;

	// Vector with the chip shifts for each correlator tap
    //GPU memory (can be mapped to CPU memory in shared memory devices!)
	hipMalloc((void **)&d_shifts_chips, sizeof(float)*n_correlators);
	hipMemset(d_shifts_chips,0,sizeof(float)*n_correlators);

	//scalars
	//hipMalloc((void **)&d_corr_out, sizeof(std::complex<float>)*n_correlators);
	//hipMemset(d_corr_out,0,sizeof(std::complex<float>)*n_correlators);

    // Launch the Vector Add CUDA Kernel
    // TODO: write a smart load balance using device info!
	threadsPerBlock = 64;
    blocksPerGrid = 128;//(int)(signal_length_samples+threadsPerBlock-1)/threadsPerBlock;

	hipStreamCreate (&stream1) ;
	//hipStreamCreate (&stream2) ;
	return true;
}

bool cuda_multicorrelator::set_local_code_and_taps(
		int code_length_chips,
		const std::complex<float>* local_codes_in,
		float *shifts_chips,
		int n_correlators
		)
{

          hipSetDevice(selected_gps_device);
	//********* ZERO COPY VERSION ************
//	// Get device pointer from host memory. No allocation or memcpy
//	hipError_t code;
//	// local code CPU -> GPU copy memory
//	code=hipHostGetDevicePointer((void **)&d_local_codes_in,  (void *) local_codes_in, 0);
//	if (code!=hipSuccess)
//	{
//		printf("cuda hipHostGetDevicePointer error in set_local_code_and_taps \r\n");
//	}
//	// Correlator shifts vector CPU -> GPU copy memory (fractional chip shifts are allowed!)
//	code=hipHostGetDevicePointer((void **)&d_shifts_chips,  (void *) shifts_chips, 0);
//	if (code!=hipSuccess)
//	{
//		printf("cuda hipHostGetDevicePointer error in set_local_code_and_taps \r\n");
//	}

	//******** CudaMalloc version ***********
    //local code CPU -> GPU copy memory
    hipMemcpyAsync(d_local_codes_in, local_codes_in, sizeof(GPU_Complex)*code_length_chips, hipMemcpyHostToDevice,stream1);
    d_code_length_chips=code_length_chips;

    //Correlator shifts vector CPU -> GPU copy memory (fractional chip shifts are allowed!)
    hipMemcpyAsync(d_shifts_chips, shifts_chips, sizeof(float)*n_correlators,
                                    hipMemcpyHostToDevice,stream1);

	return true;
}

bool cuda_multicorrelator::set_input_output_vectors(
		std::complex<float>* corr_out,
		std::complex<float>* sig_in
		)
{

         hipSetDevice(selected_gps_device);
	// Save CPU pointers
	d_sig_in_cpu =sig_in;
	d_corr_out_cpu = corr_out;

	// Zero Copy version
	// Get device pointer from host memory. No allocation or memcpy
	hipError_t code;
	code=hipHostGetDevicePointer((void **)&d_sig_in,  (void *) sig_in, 0);
	code=hipHostGetDevicePointer((void **)&d_corr_out,  (void *) corr_out, 0);
	if (code!=hipSuccess)
	{
		printf("cuda hipHostGetDevicePointer error \r\n");
	}
	return true;

}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

bool cuda_multicorrelator::Carrier_wipeoff_multicorrelator_resampler_cuda(
		float rem_carrier_phase_in_rad,
		float phase_step_rad,
        float code_phase_step_chips,
        float rem_code_phase_chips,
		int signal_length_samples,
		int n_correlators)
	{

    hipSetDevice(selected_gps_device); 
	// cudaMemCpy version
	//size_t memSize = signal_length_samples * sizeof(std::complex<float>);
	// input signal CPU -> GPU copy memory
    //hipMemcpyAsync(d_sig_in, d_sig_in_cpu, memSize,
    //                               hipMemcpyHostToDevice, stream2);
    //***** NOTICE: NCO is computed on-the-fly, not need to copy NCO into GPU! ****

    //launch the multitap correlator with integrated local code resampler!

    Doppler_wippe_scalarProdGPUCPXxN_shifts_chips<<<blocksPerGrid, threadsPerBlock,0 ,stream1>>>(
			d_corr_out,
			d_sig_in,
			d_sig_doppler_wiped,
			d_local_codes_in,
			d_shifts_chips,
			d_code_length_chips,
	        code_phase_step_chips,
	        rem_code_phase_chips,
			n_correlators,
			signal_length_samples,
			rem_carrier_phase_in_rad,
			phase_step_rad
			);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipStreamSynchronize(stream1));

	// cudaMemCpy version
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    //scalar products (correlators outputs)
    //hipMemcpyAsync(d_corr_out_cpu, d_corr_out, sizeof(std::complex<float>)*n_correlators,
    //        hipMemcpyDeviceToHost,stream1);
    return true;
}

cuda_multicorrelator::cuda_multicorrelator()
{
	d_sig_in=NULL;
	d_nco_in=NULL;
	d_sig_doppler_wiped=NULL;
	d_local_codes_in=NULL;
	d_shifts_samples=NULL;
	d_shifts_chips=NULL;
	d_corr_out=NULL;
	threadsPerBlock=0;
	blocksPerGrid=0;
	d_code_length_chips=0;
}

bool cuda_multicorrelator::free_cuda()
{
	// Free device global memory
	if (d_sig_in!=NULL) hipFree(d_sig_in);
	if (d_nco_in!=NULL) hipFree(d_nco_in);
	if (d_sig_doppler_wiped!=NULL) hipFree(d_sig_doppler_wiped);
	if (d_local_codes_in!=NULL) hipFree(d_local_codes_in);
	if (d_corr_out!=NULL) hipFree(d_corr_out);
	if (d_shifts_samples!=NULL) hipFree(d_shifts_samples);
	if (d_shifts_chips!=NULL) hipFree(d_shifts_chips);
    // Reset the device and exit
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
	hipDeviceReset();
	return true;
}

